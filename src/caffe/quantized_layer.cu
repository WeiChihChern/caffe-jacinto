#include "hip/hip_runtime.h"
#include "caffe/quantized_layer.hpp"
#include "caffe/quantized_layer.cuh"

namespace caffe {


template<typename Ftype, typename Btype>
void QuantizedLayer<Ftype, Btype>::Quantize_gpu(const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  if (this->layer_param_.has_quantization_param()) {
    //LOG(INFO) << "Quantizing layer: " << this->layer_param_.name();
    const vector<shared_ptr<Blob > >& blobs = this->blobs();
    QuantizationParameter& param = *this->layer_param_.mutable_quantization_param();
    if (param.precision() != QuantizationParameter_Precision_FLOAT) {
      // Trim layer input
      for (int i = 0; i < std::min<int>(param.qparam_in_size(),bottom.size()); ++i) {
        if(param.qparam_in(i).quantize()) {
          this->QuantizeLayerInputs_gpu(bottom[i]->mutable_gpu_data<Ftype>(), i, bottom[i]->count());
        }
      }

      // Trim weights - do it only at the start of quantization
      for(int blob_id=0; blob_id<blobs.size(); blob_id++) {
        if(param.qparam_w(blob_id).quantize() && param.quantized_infer_count() == 0) {
      	  bool clip = (blob_id == 0);
          this->QuantizeWeights_gpu(blobs[blob_id]->mutable_gpu_data<Ftype>(), blob_id, blobs[blob_id]->count(), clip);
        }
      }

      // Trim layer output
      for (int i = 0; i < top.size(); ++i) {
        if(param.qparam_out(i).quantize()) {
          this->QuantizeLayerOutputs_gpu(top[i]->mutable_gpu_data<Ftype>(), i, top[i]->count());
        }
      }
    }
  }
}


template<typename Ftype, typename Btype>
void QuantizedLayer<Ftype, Btype>::QuantizeWeights_gpu(Ftype* data, const int blob_id, const int count, bool clip) {
  const QuantizationParameter& param = this->layer_param_.quantization_param();
  const QuantizationParameter::QParams& qparam_w = param.qparam_w(blob_id);
  switch (param.precision()) {
  case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
    Trim2FixedPoint_gpu(data, count, param.power2_scale_weights(), qparam_w.bitwidth(),
        param.rounding_scheme(), qparam_w.fracbits(), qparam_w.scale_target(),
        qparam_w.offset(), qparam_w.unsigned_quant(), clip, false);
    break;
  case QuantizationParameter_Precision_FLOAT:
	  break;
  default:
    LOG(FATAL) << "Unknown trimming mode: " << param.precision() << " for layer:" << this->layer_param_.name();
    break;
  }
}


template<typename Ftype, typename Btype>
void QuantizedLayer<Ftype, Btype>::QuantizeLayerInputs_gpu(
    Ftype* data, const int blob_id, const int count) {
  const QuantizationParameter& param = this->layer_param_.quantization_param();
  const QuantizationParameter::QParams& qparam_in = param.qparam_in(blob_id);
  switch (param.precision()) {
    case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
      Trim2FixedPoint_gpu(data, count, param.power2_scale_activations(), qparam_in.bitwidth(),
          param.rounding_scheme(), qparam_in.fracbits(), qparam_in.scale_target(),
          qparam_in.offset(), qparam_in.unsigned_quant(), true, true);
      break;
    case QuantizationParameter_Precision_FLOAT:
  	  break;
    default:
      LOG(FATAL) << "Unknown trimming mode: " << param.precision() << " for layer:" << this->layer_param_.name();
      break;
  }
}

template<typename Ftype, typename Btype>
void QuantizedLayer<Ftype, Btype>::QuantizeLayerOutputs_gpu(Ftype* data,
      const int blob_id, const int count) {
  const QuantizationParameter& param = this->layer_param_.quantization_param();
  const QuantizationParameter::QParams& qparam_out = param.qparam_out(blob_id);
  switch (param.precision()) {
    case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
      Trim2FixedPoint_gpu(data, count, param.power2_scale_activations(), qparam_out.bitwidth(),
          param.rounding_scheme(), qparam_out.fracbits(), qparam_out.scale_target(),
          qparam_out.offset(), qparam_out.unsigned_quant(), true, true);
      break;
    case QuantizationParameter_Precision_FLOAT:
  	  break;
    default:
      LOG(FATAL) << "Unknown trimming mode: " << param.precision() << " for layer:" << this->layer_param_.name();
      break;
  }
}

template <typename Dtype>
__global__ void Trim2FixedPoint_roundassymetric_kernel(Dtype* data, const int cnt,
    const int bitwidth, const int rounding, float scale, float inv_scale, float offset, float min_data, float max_data, bool clip) {
    CUDA_KERNEL_LOOP(index, cnt) {

    data[index] = (data[index] * scale) + offset;

    // Round data
    switch (rounding) {
    case QuantizationParameter_Rounding_NEAREST:
      //data[index] = rint(data[index]);
      data[index] = int(data[index] >= 0? (data[index]+0.5) : (data[index]-0.5));
      break;
    case QuantizationParameter_Rounding_STOCHASTIC:
      data[index] = __float2int_rd(data[index] + RandUniform_device(index));
      break;
    default:
      break;
    }

    // Saturate data
    if(clip) {
      data[index] = (data[index]>(Dtype)max_data? (Dtype)max_data:
        (data[index]<(Dtype)min_data?(Dtype)min_data:data[index]));
    }

    data[index] = (data[index] - offset) * inv_scale;
  }
}

template <typename Dtype>
__global__ void Trim2FixedPoint_roundup_kernel(Dtype* data, const int cnt,
    const int bitwidth, const int rounding, float scale, float inv_scale, float offset, float min_data, float max_data, bool clip) {
    CUDA_KERNEL_LOOP(index, cnt) {

    data[index] = (data[index] * scale) + offset;

    // Round data
    switch (rounding) {
    case QuantizationParameter_Rounding_NEAREST:
      //data[index] = rint(data[index]);
      //data[index] = (data[index]+0.5);
      //match rounding with typical rounding done in C
      //TODO: replace 4096 with the scale_target of the data
      data[index] = (int(data[index] * 4096) + 2048)>>12;
      break;
    case QuantizationParameter_Rounding_STOCHASTIC:
      data[index] = __float2int_rd(data[index] + RandUniform_device(index));
      break;
    default:
      break;
    }

    // Saturate data
    if(clip) {
      data[index] = (data[index]>(Dtype)max_data? (Dtype)max_data:
        (data[index]<(Dtype)min_data?(Dtype)min_data:data[index]));
    }

    data[index] = (data[index] - offset) * inv_scale;
  }
}


template<typename Ftype, typename Btype>
void QuantizedLayer<Ftype, Btype>::Trim2FixedPoint_gpu(Ftype* data, const int cnt, bool power2_range,
      const int bitwidth, const int rounding, int fracbits, float scale, float offset, bool unsigned_quant, bool clip,
      bool roundup) {
  float inv_scale = 1.0f/scale;

  int qrange = unsigned_quant? bitwidth :  (bitwidth - 1);
  float min_data = unsigned_quant? 0 : -(powf(2, qrange));
  float max_data = +(powf(2, qrange) - 1);

  if(roundup) {
      Trim2FixedPoint_roundup_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(
          data, cnt, bitwidth, rounding, scale, inv_scale, offset, min_data, max_data, clip);
  } else {
      Trim2FixedPoint_roundassymetric_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(
          data, cnt, bitwidth, rounding, scale, inv_scale, offset, min_data, max_data, clip);
  }
}


template void QuantizedLayer<double, double>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);
template void QuantizedLayer<double, float>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);
template void QuantizedLayer<double, float16>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);

template void QuantizedLayer<float, double>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);
template void QuantizedLayer<float, float>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);
template void QuantizedLayer<float, float16>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);

template void QuantizedLayer<float16, double>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);
template void QuantizedLayer<float16, float>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);
template void QuantizedLayer<float16, float16>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);


}  // namespace caffe


